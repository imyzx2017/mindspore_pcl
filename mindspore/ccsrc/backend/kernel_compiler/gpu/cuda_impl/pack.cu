#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "backend/kernel_compiler/gpu/cuda_impl/pack.cuh"
template <typename T>
__global__ void Pack(const int size, const int input_num, const int dims_behind_axis, T** inputs, T* output) {
  for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
      int cycle = pos / (input_num * dims_behind_axis);
      int cur_input_index = pos % (input_num * dims_behind_axis) / dims_behind_axis;
      int local_index = pos % (input_num * dims_behind_axis) % dims_behind_axis;
      output[pos] = inputs[cur_input_index][cycle * dims_behind_axis + local_index];
  }
  return;
}

template <typename T>
void PackKernel(const int size, const int input_num,
                const int dims_behind_axis, T** inputs, T* output,
                hipStream_t hip_stream) {
  Pack<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input_num, dims_behind_axis, inputs, output);
  return;
}

template void PackKernel(const int size, const int input_num,
                           const int dims_behind_axis, float** inputs, float* output,
                           hipStream_t hip_stream);
template void PackKernel(const int size, const int input_num,
                           const int dims_behind_axis, int** inputs, int* output,
                           hipStream_t hip_stream);
template void PackKernel(const int size, const int input_num,
                           const int dims_behind_axis, half** inputs, half* output,
                           hipStream_t hip_stream);
template void PackKernel(const int size, const int input_num,
                           const int dims_behind_axis, short** inputs, short* output,  // NOLINT
                           hipStream_t hip_stream);
template void PackKernel(const int size, const int input_num,
                           const int dims_behind_axis, unsigned char** inputs, unsigned char* output,
                           hipStream_t hip_stream);
template void PackKernel(const int size, const int input_num,
                           const int dims_behind_axis, bool** inputs, bool* output,
                           hipStream_t hip_stream);
