#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include "loss_with_reduction_impl.cuh"
#include "runtime/device/gpu/cuda_common.h"

inline __device__ float logT(float x) { return logf(x); }
inline __device__ half logT(half x) { return hlog(x); }
inline __device__ float castT(float ref, int x) { return __int2float_rd(x); }
inline __device__ half castT(half ref, int x) { return __int2half_rd(x); }
inline __device__ float maxT(float a, float b) { return fmaxf(a, b); }
inline __device__ half maxT(half a, half b) { return a > b ? a : b; }

template <typename T>
__global__ void Copy(T *loss, T *tmp_loss, int reduction, int input_size) {
  loss[0] += tmp_loss[0];
  if (reduction == 1) {
    loss[0] /= castT(loss[0], input_size);
  }
}

template <typename T>
__global__ void AddTile(T *tmp_loss, int index) {
  tmp_loss[0] += tmp_loss[index];
}
template <typename T>
__global__ void PartialSum(T *tmp_loss, int stride) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < stride; i += blockDim.x * gridDim.x) {
    tmp_loss[i] += tmp_loss[i + stride];
  }
}

template <typename T>
__global__ void LossInitKernel(T *loss) {
  loss[0] = static_cast<T>(0.);
}

template <typename T>
__global__ void KLDivLossKernel(const int input_size, const int reduction, const T *input_x, const T *input_y, T *loss,
                                T *tmp_loss) {
  T epsilon = 1e-6;
  if (reduction == 0) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T denominator = maxT(input_y[i], epsilon);
      T value = input_y[i] * (logT(denominator) - input_x[i]);
      loss[i] = value;
    }
  } else {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T denominator = maxT(input_y[i], epsilon);
      T value = input_y[i] * (logT(denominator) - input_x[i]);
      tmp_loss[i] = value;
    }
  }
}

template <typename T>
void KLDivLoss(const int &input_size, const int &reduction, const T *input_x, const T *input_y, T *loss, T *tmp_loss,
               hipStream_t stream) {
  LossInitKernel<<<1, 1, 0, stream>>>(loss);
  KLDivLossKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, reduction, input_x, input_y, loss,
                                                                      tmp_loss);
  if (reduction != 0) {
    if (input_size % 2 == 1) {
      AddTile<<<1, 1, 0, stream>>>(tmp_loss, input_size - 1);
    }
    for (int stride = input_size / 2; stride > 0; stride >>= 1) {
      PartialSum<<<GET_BLOCKS(stride), GET_THREADS, 0, stream>>>(tmp_loss, stride);
      if (stride > 2 && stride % 2 == 1) {
        AddTile<<<1, 1, 0, stream>>>(tmp_loss, stride - 1);
      }
    }
    Copy<<<1, 1, 0, stream>>>(loss, tmp_loss, reduction, input_size);
  }
}

template <typename T>
__global__ void KLDivLossGradKernel(const int input_size, const int reduction, const T *input_x, const T *input_y,
                                    const T *dloss, T *dx, T *dy) {
  T epsilon = 1e-6;
  T one = static_cast<T>(1);
  if (reduction == 0) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T denominator = maxT(input_y[i], epsilon);
      dx[i] = -input_y[i] * dloss[i];
      dy[i] = (logT(denominator) + one - input_x[i]) * dloss[i];
    }
  } else {
    T dloss1 = dloss[0];
    if (reduction == 1) {
      dloss1 = dloss[0] / castT(dloss[0], input_size);
    }
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T denominator = maxT(input_y[i], epsilon);
      dx[i] = -input_y[i] * dloss1;
      dy[i] = (logT(denominator) + one - input_x[i]) * dloss1;
    }
  }
}

template <typename T>
void KLDivLossGrad(const int &input_size, const int &reduction, const T *input_x, const T *input_y, const T *dloss,
                   T *dx, T *dy, hipStream_t stream) {
  KLDivLossGradKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, reduction, input_x, input_y,
                                                                          dloss, dx, dy);
}

template <typename T>
__global__ void BinaryCrossEntropyLossKernel(const int input_size, const int reduction, const T *input_x,
                                             const T *input_y, const T *weight, T *loss, T *tmp_loss) {
  T epsilon = 1e-12;
  T one = static_cast<T>(1);
  if (reduction == 0) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T value =
        -weight[i] * (input_y[i] * logT(input_x[i] + epsilon) + (one - input_y[i]) * logT(one - input_x[i] + epsilon));
      loss[i] = value;
    }
  } else {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T value =
        -weight[i] * (input_y[i] * logT(input_x[i] + epsilon) + (one - input_y[i]) * logT(one - input_x[i] + epsilon));
      tmp_loss[i] = value;
    }
  }
}

template <typename T>
void BinaryCrossEntropyLoss(const int &input_size, const int &reduction, const T *input_x, const T *input_y,
                            const T *weight, T *loss, T *tmp_loss, hipStream_t stream) {
  LossInitKernel<<<1, 1, 0, stream>>>(loss);
  BinaryCrossEntropyLossKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, reduction, input_x,
                                                                                   input_y, weight, loss, tmp_loss);
  if (reduction != 0) {
    if (input_size % 2 == 1) {
      AddTile<<<1, 1, 0, stream>>>(tmp_loss, input_size - 1);
    }
    for (int stride = input_size / 2; stride > 0; stride >>= 1) {
      PartialSum<<<GET_BLOCKS(stride), GET_THREADS, 0, stream>>>(tmp_loss, stride);
      if (stride > 2 && stride % 2 == 1) {
        AddTile<<<1, 1, 0, stream>>>(tmp_loss, stride - 1);
      }
    }
    Copy<<<1, 1, 0, stream>>>(loss, tmp_loss, reduction, input_size);
  }
}

template <typename T>
__global__ void BinaryCrossEntropyLossGradKernel(const int input_size, const int reduction, const T *input_x,
                                                 const T *input_y, const T *weight, const T *dloss, T *dx) {
  T epsilon = 1e-12;
  T one = static_cast<T>(1);
  if (reduction == 0) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T denominator = maxT(input_x[i] * (one - input_x[i]), epsilon);
      T value = weight[i] * (input_x[i] - input_y[i]) / denominator;
      dx[i] = value * dloss[i];
    }
  } else {
    T dloss1 = dloss[0];
    if (reduction == 1) {
      dloss1 = dloss[0] / castT(dloss[0], input_size);
    }
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
      T denominator = maxT(input_x[i] * (one - input_x[i]), epsilon);
      T value = weight[i] * (input_x[i] - input_y[i]) / denominator;
      dx[i] = value * dloss1;
    }
  }
}

template <typename T>
void BinaryCrossEntropyLossGrad(const int &input_size, const int &reduction, const T *input_x, const T *input_y,
                                const T *weight, const T *dloss, T *dx, hipStream_t stream) {
  BinaryCrossEntropyLossGradKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, reduction, input_x,
                                                                                       input_y, weight, dloss, dx);
}

template void KLDivLoss<float>(const int &input_size, const int &reduction, const float *input_x, const float *input_y,
                               float *loss, float *tmp_loss, hipStream_t stream);

template void KLDivLossGrad<float>(const int &input_size, const int &reduction, const float *input_x,
                                   const float *input_y, const float *dloss, float *dx, float *dy, hipStream_t stream);

template void BinaryCrossEntropyLoss<float>(const int &input_size, const int &reduction, const float *input_x,
                                            const float *input_y, const float *weight, float *loss, float *tmp_loss,
                                            hipStream_t stream);

template void BinaryCrossEntropyLossGrad<float>(const int &input_size, const int &reduction, const float *input_x,
                                                const float *input_y, const float *weight, const float *dloss,
                                                float *dx, hipStream_t stream);

template void KLDivLoss<half>(const int &input_size, const int &reduction, const half *input_x, const half *input_y,
                              half *loss, half *tmp_loss, hipStream_t stream);

template void KLDivLossGrad<half>(const int &input_size, const int &reduction, const half *input_x, const half *input_y,
                                  const half *dloss, half *dx, half *dy, hipStream_t stream);

template void BinaryCrossEntropyLoss<half>(const int &input_size, const int &reduction, const half *input_x,
                                           const half *input_y, const half *weight, half *loss, half *tmp_loss,
                                           hipStream_t stream);

template void BinaryCrossEntropyLossGrad<half>(const int &input_size, const int &reduction, const half *input_x,
                                               const half *input_y, const half *weight, const half *dloss, half *dx,
                                               hipStream_t stream);
