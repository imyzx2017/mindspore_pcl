#include "hip/hip_runtime.h"
/**
 * Copyright 2019 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include "backend/kernel_compiler/gpu/cuda_impl/pad_impl.cuh"

// For internal OP use, not user facing
template <typename T>
__global__ void Pad(const size_t size, const T* input, const int num, const int channels, const int old_height,
                    const int old_width, const int padded_height, const int padded_width, const int pad_top,
                    const int pad_left, float pad_value, T* output) {
  T pad_value_ = static_cast<T>(pad_value);
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    int block_num = pos / padded_width / padded_height;
    const int padded_w = pos % padded_width;
    const int padded_h = pos / padded_width % padded_height;
    if (padded_h - pad_top < 0 || padded_w - pad_left < 0 || padded_h - pad_top >= old_height ||
        padded_w - pad_left >= old_width) {
      output[pos] = pad_value_;
    } else {
      output[pos] = input[(block_num * old_height + padded_h - pad_top) * old_width + padded_w - pad_left];
    }
  }
  return;
}

// For internal OP use, not user facing
template <typename T>
__global__ void PadNHWC(const size_t size, const T* input, const int num, const int old_height, const int old_width,
                        const int channels, const int padded_height, const int padded_width, const int pad_top,
                        const int pad_left, float pad_value, T* output) {
  T pad_value_ = static_cast<T>(pad_value);
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    int block_num = pos / channels / padded_width / padded_height;
    const int padded_w = pos / channels % padded_width;
    const int padded_h = pos / channels / padded_width % padded_height;
    if (padded_h - pad_top < 0 || padded_w - pad_left < 0 || padded_h - pad_top >= old_height ||
        padded_w - pad_left >= old_width) {
      output[pos] = pad_value_;
    } else {
      output[pos] = input[((block_num * old_height + padded_h - pad_top) * old_width + padded_w - pad_left)
                            *channels + pos % channels];
    }
  }
  return;
}

// Used by user facing 'Pad' API
template <typename T>
__global__ void PadGeneral(const size_t size, const T *input, const int num, const int channels_orig,
                           const int pad_channel_before, const int pad_channel_after, const int old_height,
                           const int old_width, const int padded_height, const int padded_width, const int pad_top,
                           const int pad_left, float pad_value, T *output) {
  T pad_value_template = static_cast<T>(pad_value);
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    int block_num = (pos / padded_width) / padded_height;       // total blocks = (batch * channels)
    const int padded_w = pos % padded_width;                  // x coordinate refered to by cur 'pos'
    const int padded_h = (pos / padded_width) % padded_height;  // y coordinate refered to by cur 'pos'

    int channels_new = channels_orig + pad_channel_after + pad_channel_before;  // new number of channels from padding
    int channel_num = block_num % channels_new;                                 // current channel
    int batch_item = block_num / channels_new;                                  // current item in batch
    int equiv_block_num = 0;  // init variable to select equivalent block to copy data from from input

    if (padded_h - pad_top < 0 || padded_w - pad_left < 0 || padded_h - pad_top >= old_height ||
        padded_w - pad_left >= old_width || channel_num <= pad_channel_before - 1 ||
        channel_num > channels_orig + pad_channel_before - 1) {
      output[pos] = pad_value_template;
    } else {
      // on a block/x,y positon that isn't padding, copy data from the correct block/x,y pos the input
      // calculate from number of blocks of padding (due to channel padding) inserted prior
      equiv_block_num = block_num - (batch_item * (pad_channel_before + pad_channel_after)) - pad_channel_before;
      output[pos] = input[(equiv_block_num * old_height + padded_h - pad_top) * old_width + padded_w - pad_left];
    }
  }
  return;
}

template <typename T>
__global__ void PadGradNHWC(const size_t size, const T* dy, const int num, const int old_height, const int old_width,
                        const int channels, const int padded_height, const int padded_width, const int pad_top,
                        const int pad_left, T* dx) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    int block_num = pos / channels / old_width / old_height;
    const int padded_w = pos / channels % old_width + pad_left;
    const int padded_h = pos / channels / old_width % old_height + pad_top;
    dx[pos] = dy[((block_num * padded_height + padded_h) * padded_width + padded_w)*channels+pos%channels];
  }
  return;
}

template <typename T>
__global__ void PadGrad(const size_t size, const T* dy, const int num, const int channels, const int old_height,
                        const int old_width, const int padded_height, const int padded_width, const int pad_top,
                        const int pad_left, T* dx) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    int block_num = pos / old_width / old_height;
    const int padded_w = pos % old_width + pad_left;
    const int padded_h = pos / old_width % old_height + pad_top;
    dx[pos] = dy[(block_num * padded_height + padded_h) * padded_width + padded_w];
  }
  return;
}

template <typename T>
void CalPad(const size_t size, const T* input, const int num, const int channels, const int old_height,
            const int old_width, const int padded_height, const int padded_width, const int pad_top, const int pad_left,
            const float pad_value, T* output, hipStream_t hip_stream) {
  Pad<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input, num, channels, old_height, old_width,
                                                         padded_height, padded_width, pad_top, pad_left, pad_value,
                                                         output);
  return;
}

template <typename T>
void CalPadNHWC(const size_t size, const T* input, const int num, const int old_height, const int old_width,
                const int channels, const int padded_height, const int padded_width, const int pad_top,
                const int pad_left, const float pad_value, T* output, hipStream_t hip_stream) {
  PadNHWC<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input, num, old_height, old_width, channels,
      padded_height, padded_width, pad_top, pad_left, pad_value, output);
  return;
}

template <typename T>
void CalPadGeneral(const size_t size, const T *input, const int num, const int channels_orig,
                   const int pad_channel_before, const int pad_channel_after, const int old_height, const int old_width,
                   const int padded_height, const int padded_width, const int pad_top, const int pad_left,
                   float pad_value, T *output, hipStream_t hip_stream) {
  PadGeneral<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input, num, channels_orig, pad_channel_before,
                                                                pad_channel_after, old_height, old_width, padded_height,
                                                                padded_width, pad_top, pad_left, pad_value, output);
  return;
}

template <typename T>
void CalPadGradNHWC(const size_t size, const T* dy, const int num, const int old_height, const int old_width,
                 const int channels, const int padded_height, const int padded_width, const int pad_top,
                const int pad_left, T* dx, hipStream_t hip_stream) {
  PadGradNHWC<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, dy, num, old_height, old_width, channels,
      padded_height, padded_width, pad_top, pad_left, dx);
  return;
}

template <typename T>
void CalPadGrad(const size_t size, const T* dy, const int num, const int channels, const int old_height,
                const int old_width, const int padded_height, const int padded_width, const int pad_top,
                const int pad_left, T* dx, hipStream_t hip_stream) {
  PadGrad<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, dy, num, channels, old_height, old_width,
                                                             padded_height, padded_width, pad_top, pad_left, dx);
  return;
}

template void CalPad<float>(const size_t size, const float* input, const int num, const int channels,
                            const int old_height, const int old_width, const int padded_height, const int padded_width,
                            const int pad_top, const int pad_left, float pad_value, float* output,
                            hipStream_t hip_stream);
template void CalPadGrad<float>(const size_t size, const float* dy, const int num, const int channels,
                                const int old_height, const int old_width, const int padded_height,
                                const int padded_width, const int pad_top, const int pad_left, float* dx,
                                hipStream_t hip_stream);
template void CalPad<half>(const size_t size, const half* input, const int num, const int channels,
                           const int old_height, const int old_width, const int padded_height, const int padded_width,
                           const int pad_top, const int pad_left, float pad_value, half* output,
                           hipStream_t hip_stream);
template void CalPadGrad<half>(const size_t size, const half* dy, const int num, const int channels,
                               const int old_height, const int old_width, const int padded_height,
                               const int padded_width, const int pad_top, const int pad_left, half* dx,
                               hipStream_t hip_stream);
template void CalPadNHWC<float>(const size_t size, const float* input, const int num, const int old_height,
                                const int old_width, const int channels, const int padded_height,
                                const int padded_width, const int pad_top, const int pad_left, float pad_value,
                                float* output, hipStream_t hip_stream);
template void CalPadNHWC<half>(const size_t size, const half* input, const int num, const int old_height,
                               const int old_width, const int channels, const int padded_height,
                               const int padded_width, const int pad_top, const int pad_left, float pad_value,
                               half* output, hipStream_t hip_stream);
template void CalPadGradNHWC<float>(const size_t size, const float* dy, const int num, const int old_height,
                                    const int old_width, const int channels, const int padded_height,
                                    const int padded_width, const int pad_top, const int pad_left, float* dx,
                                    hipStream_t hip_stream);
template void CalPadGradNHWC<half>(const size_t size, const half* dy, const int num, const int old_height,
                                   const int old_width, const int channels, const int padded_height,
                                   const int padded_width, const int pad_top, const int pad_left, half* dx,
                                   hipStream_t hip_stream);
template void CalPadGeneral<float>(const size_t size, const float *input, const int num, const int channels_orig,
                                   const int pad_channel_before, const int pad_channel_after, const int old_height,
                                   const int old_width, const int padded_height, const int padded_width,
                                   const int pad_top, const int pad_left, float pad_value, float *output,
                                   hipStream_t hip_stream);
template void CalPadGeneral<half>(const size_t size, const half *input, const int num, const int channels_orig,
                                  const int pad_channel_before, const int pad_channel_after, const int old_height,
                                  const int old_width, const int padded_height, const int padded_width,
                                  const int pad_top, const int pad_left, float pad_value, half *output,
                                  hipStream_t hip_stream);
template void CalPadGeneral<int>(const size_t size, const int *input, const int num, const int channels_orig,
                                  const int pad_channel_before, const int pad_channel_after, const int old_height,
                                  const int old_width, const int padded_height, const int padded_width,
                                  const int pad_top, const int pad_left, float pad_value, int *output,
                                  hipStream_t hip_stream);
