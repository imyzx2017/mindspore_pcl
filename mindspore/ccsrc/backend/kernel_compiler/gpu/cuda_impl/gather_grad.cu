#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iostream>
#include "backend/kernel_compiler/gpu/cuda_impl/gather_grad.cuh"
#include "runtime/device/gpu/cuda_common.h"
template <typename T, typename S>
__global__ void GatherGradKernel(const T *index, const S *grad, S *output, const size_t output_dim0,
                                 const size_t output_dim1, const size_t output_dim2) {
  size_t num = output_dim0 * output_dim1 * output_dim2;
  size_t i, k;
  for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < num;
       id += blockDim.x * gridDim.x) {
    i = id / (output_dim1 * output_dim2) % output_dim0;
    k = id % output_dim2;

    size_t j_read = static_cast<size_t>(index[id]);
    size_t read_id = i * output_dim1 * output_dim2 + j_read * output_dim2 + k;
    output[read_id] = grad[id];
  }
  return;
}
template <typename T, typename S>
void GatherGrad(const T *index, const S *grad, S *output, const size_t output_dim0,
                const size_t output_dim1, const size_t output_dim2, hipStream_t stream) {
  size_t size = output_dim0 * output_dim1 * output_dim2;
  GatherGradKernel<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(index, grad, output,
                                                                 output_dim0, output_dim1, output_dim2);
  return;
}

template void GatherGrad<int, float>(const int *index, const float *grad, float *output,
                                     const size_t output_dim0, const size_t output_dim1,
                                     const size_t output_dim2, hipStream_t stream);

template void GatherGrad<int, half>(const int *index, const half *grad, half *output,
                                    const size_t output_dim0, const size_t output_dim1,
                                    const size_t output_dim2, hipStream_t stream);

template void GatherGrad<int64_t, float>(const int64_t *index, const float *grad, float *output,
                                         const size_t output_dim0, const size_t output_dim1,
                                         const size_t output_dim2, hipStream_t stream);

template void GatherGrad<int64_t, half>(const int64_t *index, const half *grad, half *output,
                                        const size_t output_dim0, const size_t output_dim1,
                                        const size_t output_dim2, hipStream_t stream);
