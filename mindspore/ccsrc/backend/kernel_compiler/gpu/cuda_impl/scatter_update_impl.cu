#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "backend/kernel_compiler/gpu/cuda_impl/scatter_update_impl.cuh"

template <typename T>
__global__ void ScatterUpdate(const int inner_size, const int updates_size, const int *indices, const T *updates,
                              T *output) {
  for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < updates_size; pos += blockDim.x * gridDim.x) {
    const int index = pos / inner_size;
    const int offset = pos % inner_size;
    const int current_pos = indices[index] * inner_size + offset;
    output[current_pos] = updates[pos];
  }
}

template <typename T>
void CalScatterUpdate(const int &inner_size, const int &indices_size, const int *indices, const T *updates, T *output,
                      hipStream_t hip_stream) {
  const int updates_size = inner_size * indices_size;
  ScatterUpdate<<<GET_BLOCKS(updates_size), GET_THREADS, 0, hip_stream>>>(inner_size, updates_size, indices, updates,
                                                                           output);
}

template void CalScatterUpdate<float>(const int &inner_size, const int &indices_size, const int *indices,
                                      const float *updates, float *output, hipStream_t hip_stream);
template void CalScatterUpdate<half>(const int &inner_size, const int &indices_size, const int *indices,
                                     const half *updates, half *output, hipStream_t hip_stream);
template void CalScatterUpdate<int>(const int &inner_size, const int &indices_size, const int *indices,
                                    const int *updates, int *output, hipStream_t hip_stream);
