#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "backend/kernel_compiler/gpu/cuda_impl/sigmoid_cross_entropy_with_logits_impl.cuh"

template <typename T, typename S>
__global__ void SigmoidCrossEntropyWithLogitsKernel(const size_t size, const T *logits, const S *labels, T *outputs) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    const T reverse_factor = static_cast<T>(logits[i] >= 0);
    outputs[i] = log1p(exp(logits[i] - 2 * reverse_factor * logits[i])) - logits[i] * (labels[i] - reverse_factor);
  }
}

template <typename T, typename S>
void SigmoidCrossEntropyWithLogits(const size_t size, const T *logits, const S *labels, T *outputs,
                                   hipStream_t hip_stream) {
  SigmoidCrossEntropyWithLogitsKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, logits, labels, outputs);
}

template void SigmoidCrossEntropyWithLogits<float, float>(const size_t size, const float *logits, const float *labels,
                                                          float *outputs, hipStream_t hip_stream);
