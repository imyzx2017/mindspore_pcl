#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "unary_op_grad_impl.cuh"

template <typename T>
__global__ void SqrtGradKernel(const T *input, const T *dout, T *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    float input_f = static_cast<float>(input[i]);
    float dout_f = static_cast<float>(dout[i]);
    float res_vmul = dout_f / (2.0 * input_f);
    output[i] = static_cast<T>(res_vmul);
  }
  return;
}
template <typename T>
__global__ void RsqrtGradKernel(const T *input, const T *dout, T *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    float input_f = static_cast<float>(input[i]);
    float dout_f = static_cast<float>(dout[i]);
    float res_vmul = input_f * input_f * input_f;
    res_vmul = -0.5 * res_vmul * dout_f;
    output[i] = static_cast<T>(res_vmul);
  }
  return;
}
template <typename T>
__global__ void AsinGradKernel(const T *input, const T *dout, T *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    T one = 1;
    T sqt = sqrtf(one - input[i] * input[i]);
    output[i] = dout[i] / sqt;
  }
  return;
}
template <>
__global__ void AsinGradKernel(const half *input, const half *dout, half *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    half one = 1;
    half sqt = hsqrt(one - input[i] * input[i]);
    output[i] = dout[i] / sqt;
  }
  return;
}
template <typename T>
__global__ void ACosGradKernel(const T *input, const T *dout, T *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    T neg_one = -1;
    T one = 1;
    T sqt = sqrtf(one - input[i] * input[i]);
    output[i] = neg_one * dout[i] / sqt;
  }
  return;
}
template <>
__global__ void ACosGradKernel(const half *input, const half *dout, half *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    half neg_one = -1;
    half one = 1;
    half sqt = hsqrt(one - input[i] * input[i]);
    output[i] = neg_one * dout[i] / sqt;
  }
  return;
}
template <typename T>
void SqrtGrad(const T *input, const T *dout, T *output, const size_t count, hipStream_t hip_stream) {
  SqrtGradKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, dout, output, count);
  return;
}
template <typename T>
void RsqrtGrad(const T *input, const T *dout, T *output, const size_t count, hipStream_t hip_stream) {
  RsqrtGradKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, dout, output, count);
  return;
}

template <typename T>
void AsinGrad(const T *input, const T *dout, T *output, const size_t count, hipStream_t hip_stream) {
  AsinGradKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, dout, output, count);
  return;
}

template <typename T>
void ACosGrad(const T *input, const T *dout, T *output, const size_t count, hipStream_t hip_stream) {
  ACosGradKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, dout, output, count);
  return;
}

template void SqrtGrad<float>(const float *input, const float *dout, float *output, const size_t count,
                              hipStream_t hip_stream);
template void RsqrtGrad<float>(const float *input, const float *dout, float *output, const size_t count,
                               hipStream_t hip_stream);
template void AsinGrad<float>(const float *input, const float *dout, float *output, const size_t count,
                               hipStream_t hip_stream);
template void ACosGrad<float>(const float *input, const float *dout, float *output, const size_t count,
                               hipStream_t hip_stream);
template void SqrtGrad<half>(const half *input, const half *dout, half *output, const size_t count,
                             hipStream_t hip_stream);
template void RsqrtGrad<half>(const half *input, const half *dout, half *output, const size_t count,
                              hipStream_t hip_stream);
template void AsinGrad<half>(const half *input, const half *dout, half *output, const size_t count,
                             hipStream_t hip_stream);
template void ACosGrad<half>(const half *input, const half *dout, half *output, const size_t count,
                              hipStream_t hip_stream);
