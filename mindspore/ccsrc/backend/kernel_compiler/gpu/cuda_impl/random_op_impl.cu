#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "random_op_impl.cuh"
template <typename T>
__global__ void NormalKernel(int seed, hiprandState *globalState, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    hiprand_init(seed, i, 0, &globalState[i]);
    output[i] = (T)hiprand_normal(&globalState[i]);
  }
  return;
}

template <typename T>
__global__ void UniformIntKernel(int seed, hiprandState *globalState, T *input1, size_t input_size_1,
                                 T *input2, size_t input_size_2, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    hiprand_init(seed, i, 0, &globalState[i]);
    output[i] = (T)(hiprand_uniform(&globalState[i]) * (input2[0] - input1[0])) + input1[0];
  }
  return;
}

template <typename T>
__global__ void UniformRealKernel(int seed, hiprandState *globalState, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    hiprand_init(seed, i, 0, &globalState[i]);
    output[i] = (T)hiprand_uniform(&globalState[i]);
  }
  return;
}

template <typename T>
void StandardNormal(int seed, int seed2, hiprandState *globalState, T *output, size_t count, hipStream_t hip_stream) {
  int RNG_seed = 0;
  std::random_device rd;
  if (seed2 != 0) {
    RNG_seed = seed2;
  } else if (seed != 0) {
    RNG_seed = seed;
  } else {
    RNG_seed = static_cast<int>(rd());
  }
  NormalKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(RNG_seed, globalState, output, count);
  return;
}

template <typename T>
void UniformInt(int seed, int seed2, hiprandState *globalState, T *input1, size_t input_size_1,
                T *input2, size_t input_size_2, T *output, size_t count, hipStream_t hip_stream) {
  int RNG_seed = 0;
  std::random_device rd;
  if (seed2 != 0) {
    RNG_seed = seed2;
  } else if (seed != 0) {
    RNG_seed = seed;
  } else {
    RNG_seed = static_cast<int>(rd());
  }
  UniformIntKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>
               (RNG_seed, globalState, input1, input_size_1, input2, input_size_2, output, count);
  return;
}

template <typename T>
void UniformReal(int seed, int seed2, hiprandState *globalState, T *output, size_t count, hipStream_t hip_stream) {
  int RNG_seed = 0;
  std::random_device rd;
  if (seed2 != 0) {
    RNG_seed = seed2;
  } else if (seed != 0) {
    RNG_seed = seed;
  } else {
    RNG_seed = static_cast<int>(rd());
  }
  UniformRealKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(RNG_seed, globalState, output, count);
  return;
}

template void StandardNormal<float>(int seed, int seed2, hiprandState *globalState,
                                    float *output, size_t count, hipStream_t hip_stream);
template void StandardNormal<int>(int seed, int seed2, hiprandState *globalState,
                                  int *output, size_t count, hipStream_t hip_stream);
template void UniformInt<float>(int seed, int seed2, hiprandState *globalState, float *input1, size_t input_size_1,
                                float *input2, size_t input_size_2, float *output, size_t count,
                              hipStream_t hip_stream);
template void UniformInt<int>(int seed, int seed2, hiprandState *globalState, int *input1, size_t input_size_1,
                              int *input2, size_t input_size_2, int *output, size_t count,
                              hipStream_t hip_stream);
template void UniformReal<float>(int seed, int seed2, hiprandState *globalState,
                                 float *output, size_t count, hipStream_t hip_stream);
template void UniformReal<int>(int seed, int seed2, hiprandState *globalState,
                               int *output, size_t count, hipStream_t hip_stream);
